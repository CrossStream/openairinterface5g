#include "hip/hip_runtime.h"
/*! \file slot_fep_ul.cu
 * \dft/idft feature of CUDA version
 * \author Shi-Yi Ou
 * \date 2017
 * \version 0.1
 * \company isIP LAB, Department of Computer Science, National Chiao Tung University, Hsinchu, Taiwan
 * \email: 
 * \note
 * \warning
 */
#include "stdio.h"
#include "hipfft/hipfft.h"
#include "defs.h"
#include "PHY/CUDA/extern.h"

typedef float2 Complex;

__global__ void k_rmcp(int16_t *x, Complex *y, int CP, int CP0)
{
  int i= blockDim.x * blockIdx.x+ threadIdx.x ;
  int j= (blockDim.x+CP )* blockIdx.x+ threadIdx.x + CP0;
  if (blockIdx.x > 6)
    j = j + CP0-CP;
  y[i].x = ( float )x[(j<<1)];
  y[i].y = ( float )x[(j<<1)+1];
}

__global__ void k_short(Complex *x, short *y)
{
  int i= blockDim.x * blockIdx.x+ threadIdx.x;
  y[i<<1] =   ( short )(x[i].x*0.04419417);
  y[(i<<1)+1]=( short )(x[i].y*0.04419417);
}

__global__ void k_adcp_extend( short *x, Complex *y )
{
  int i= blockDim.x * blockIdx.x+ threadIdx.x;
  y[i].x = ( float )x[ (i<<1) ];
  y[i].y = ( float )x[ (i<<1)+ 1 ];
}


void idft512ad_cu( int16_t *x, int16_t *y, int sf )
{//dl_cu
  int i;
//  printf("[CUDA] IN idft, sf num = %2d\n",sf);
  hipMemcpyAsync( dl_cu[sf].d_txdata, 
                   x,
 				   sizeof(short)* 2 * dl_cu[sf].ifftsize* dl_cu[sf].symbols_per_tti,
				   hipMemcpyHostToDevice,
				   dl_cu[sf].stream_dl );
				   
  k_adcp_extend<<< dl_cu[sf].symbols_per_tti, dl_cu[sf].ifftsize, 0, dl_cu[sf].stream_dl >>>
               ( dl_cu[sf].d_txdata,
     			 dl_cu[sf].d_txdata_ifft );
			   
  hipfftExecC2C( dl_cu[sf].ifft,
               (hipfftComplex *) dl_cu[sf].d_txdata_ifft,
			   (hipfftComplex *) dl_cu[sf].d_txdata_ifft,
 			   HIPFFT_BACKWARD);
			   
  k_short      <<< dl_cu[sf].symbols_per_tti, dl_cu[sf].ifftsize, 0, dl_cu[sf].stream_dl >>>
              ( dl_cu[sf].d_txdata_ifft,
        	    dl_cu[sf].d_txdata );  
				
  hipMemcpyAsync( dl_cu[sf].h_txdata,
                   dl_cu[sf].d_txdata,
 				   sizeof( short )* 2 * dl_cu[sf].ifftsize* dl_cu[sf].symbols_per_tti, 
				   hipMemcpyDeviceToHost,
				   dl_cu[sf].stream_dl);
  
  int index = 0;
  short *temp = dl_cu[sf].h_txdata;
  hipStreamSynchronize( dl_cu[sf].stream_dl );
  for ( i = 0; i < dl_cu[sf].symbols_per_tti; i++ )
  {
    int cp = 0;
    if( i == 0 || i == 7 )
      cp = dl_cu[sf].CP0;
    else
      cp = dl_cu[sf].CP;
    memcpy( &y[ index<<1 ], &temp[ (i+1)*dl_cu[sf].ifftsize*2-cp*2 ], cp*sizeof(short)*2 );
    memcpy( &y[ (index+cp)<<1 ], &temp[ i*dl_cu[sf].ifftsize*2 ], dl_cu[sf].ifftsize*2*sizeof(short) );
    index = index + cp + dl_cu[sf].ifftsize;
  }
}


void dft512rm_cu( int16_t *x, int16_t *y, int sf )
{
  //printf("[CUDA] symbols_per_tti = %2d samples_per_tti = %4d, fftsize = %3d, CP0 = %2d, CP= %2d\n", ul_cu[sf].symbols_per_tti,ul_cu[sf].samples_per_tti,ul_cu[sf].fftsize,ul_cu[sf].CP0,ul_cu[sf].CP);
  hipMemcpyAsync(ul_cu[sf].d_rxdata[0], 
                  x,
				  sizeof(short)* ul_cu[sf].samples_per_tti*2,
				  hipMemcpyHostToDevice,
				  ul_cu[sf].stream_ul );
  
  k_rmcp<<< ul_cu[sf].symbols_per_tti,
            ul_cu[sf].fftsize,
			0,
			ul_cu[sf].stream_ul>>>
			((short*)ul_cu[sf].d_rxdata[0], 
			 ul_cu[sf].d_rxdata_fft[0],
			 ul_cu[sf].CP,
			 ul_cu[sf].CP0);
  
  hipfftExecC2C(ul_cu[sf].fft, 
               (hipfftComplex *)ul_cu[sf].d_rxdata_fft[0],
	       (hipfftComplex *)ul_cu[sf].d_rxdata_fft[0],
	       HIPFFT_FORWARD);
  
  k_short<<< ul_cu[sf].symbols_per_tti,
             ul_cu[sf].fftsize,
			 0,
			 ul_cu[sf].stream_ul>>>
			 ( ul_cu[sf].d_rxdata_fft[0],
   			   (short *)ul_cu[sf].d_rxdataF[0]);
  
  hipMemcpyAsync(y,
                  ul_cu[sf].d_rxdataF[0], 
				  sizeof(short)* ul_cu[sf].symbols_per_tti* 2* ul_cu[sf].fftsize, 
				  hipMemcpyDeviceToHost, 
				  ul_cu[sf].stream_ul );
  hipStreamSynchronize( ul_cu[sf].stream_ul);
}














