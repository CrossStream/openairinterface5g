#include "hip/hip_runtime.h"
#include "defs.h"
#include "PHY/CUDA/extern.h"
#include "PHY/LTE_TRANSPORT/extern.h"

#include <stdio.h>


int device_count;
dl_cu_t dl_cu[10];
ul_cu_t ul_cu[10];

void init_cufft( void );
void free_cufft( void );
void init_cuda(PHY_VARS_eNB *phy_vars_eNB, LTE_DL_FRAME_PARMS frame_parms )
{
  
  int i,j,k;
  int u,v,Msc_RS;
  hipGetDeviceCount(&device_count);
  printf("[CUDA] now we have %d device\n",device_count);
  LTE_DL_FRAME_PARMS* const frame_parm = &phy_vars_eNB->lte_frame_parms;
  LTE_eNB_COMMON* const eNB_common_vars = &phy_vars_eNB->lte_eNB_common_vars;
  LTE_eNB_PUSCH** const eNB_pusch_vars  = phy_vars_eNB->lte_eNB_pusch_vars;
  LTE_eNB_SRS* const eNB_srs_vars       = phy_vars_eNB->lte_eNB_srs_vars;
  LTE_eNB_PRACH* const eNB_prach_vars   = &phy_vars_eNB->lte_eNB_prach_vars;
  for ( i = 0; i < device_count; i++ )
  {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, i);
    printf("[CUDA] device number= %d, device name= %s\n",i, deviceProp.name);
  }
  
  for ( i = 0; i < 10; i++ )
  {
    ul_cu[i].CP = frame_parms.nb_prefix_samples;
    ul_cu[i].CP0= frame_parms.nb_prefix_samples0;
	
	ul_cu[i].fftsize = frame_parms.ofdm_symbol_size;
	ul_cu[i].Ncp = frame_parms.Ncp;
	ul_cu[i].symbols_per_tti         = frame_parms.symbols_per_tti;
	ul_cu[i].samples_per_tti         = frame_parms.samples_per_tti;
	ul_cu[i].nb_antennas_rx          = frame_parms.nb_antennas_rx;
	ul_cu[i].N_RB_UL                 = frame_parms.N_RB_UL;
	
	ul_cu[i].d_rxdata                = ( int **)malloc( frame_parms.nb_antennas_rx * sizeof( int *) );
	ul_cu[i].d_rxdata_fft            = ( float2 **)malloc( frame_parms.nb_antennas_rx * sizeof( float2 *) );
	ul_cu[i].d_rxdataF               = ( int **)malloc( frame_parms.nb_antennas_rx * sizeof( int *) );
	for ( k = 0; k < ul_cu[i].nb_antennas_rx; k++ )
	{
	  if(hipMalloc(( void **)&ul_cu[i].d_rxdata[k]         , sizeof( int )* frame_parms.samples_per_tti))
            printf("error\n");
	  hipMalloc(( void **)&ul_cu[i].d_rxdata_fft[k]     , sizeof( float2 )* ul_cu[i].symbols_per_tti* ul_cu[i].fftsize);
	  hipMalloc(( void **)&ul_cu[i].d_rxdataF[k]        , 2* sizeof( int )* ul_cu[i].symbols_per_tti* ul_cu[i].fftsize );
	}
	
    dl_cu[i].CP = frame_parms.nb_prefix_samples;
    dl_cu[i].CP0= frame_parms.nb_prefix_samples0;
	
	dl_cu[i].ifftsize = frame_parms.ofdm_symbol_size;
	dl_cu[i].Ncp = frame_parms.Ncp;
	dl_cu[i].symbols_per_tti = frame_parms.symbols_per_tti;
	dl_cu[i].samples_per_tti = frame_parms.samples_per_tti;
  }
  printf("[CUDA] CP0=%d, CP=%d, fftsize=%d, symbols_per_tti=%d, samples_per_tti=%d\n",ul_cu[i].CP0,ul_cu[i].CP,frame_parms.ofdm_symbol_size,frame_parms.symbols_per_tti,frame_parms.samples_per_tti);

  init_cufft( );
  
}

void init_cufft( void )
{
  int i,j;
  short fftsize = ul_cu[i].fftsize; 
  short Ncp = ul_cu[i].Ncp; 
  short symbols_per_tti = ul_cu[i].symbols_per_tti; 
  short samples_per_tti = ul_cu[i].samples_per_tti;
  for ( i = 0; i < 10; i++ )
  {
  //for ul cuda
    hipStreamCreateWithFlags( &( ul_cu[i].stream_ul ), hipStreamNonBlocking );
	
	hipfftPlan1d( &( ul_cu[i].fft ) , fftsize ,HIPFFT_C2C, symbols_per_tti);
	hipfftSetStream( ul_cu[i].fft , ul_cu[i].stream_ul );
	
  //for dl cuda
    hipStreamCreateWithFlags( &( dl_cu[i].stream_dl ), hipStreamNonBlocking );
	hipfftPlan1d( &( dl_cu[i].ifft ) , fftsize ,HIPFFT_C2C, symbols_per_tti);
	hipMalloc((void **)&(dl_cu[i].d_txdata)     , sizeof( short )*(symbols_per_tti+1)* 2* symbols_per_tti*fftsize);
    hipMalloc((void **)&(dl_cu[i].d_txdata_o)   , sizeof( short )* samples_per_tti* 2 );
    hipMalloc((void **)&(dl_cu[i].d_txdata_ifft), sizeof( float2 )* symbols_per_tti* fftsize);
	hipHostMalloc((void **)&(dl_cu[i].h_txdata) , sizeof( short )* symbols_per_tti* 2* fftsize);  
    hipfftSetStream( dl_cu[i].ifft , dl_cu[i].stream_dl );	
  }
}

void free_cuda(void)
{
  int i, j, k;
  for ( i = 0; i < 10; i++ )
  {
    hipFree(ul_cu[i].d_rxdata);
	hipFree(ul_cu[i].d_rxdata_fft);
	hipfftDestroy(ul_cu[i].fft);
	for ( k = 0; k < ul_cu[i].nb_antennas_rx; k++ )
	{
	  hipFree(ul_cu[i].d_rxdataF[k]);
	  hipFree(ul_cu[i].d_rxdata[k]);
	  hipFree(ul_cu[i].d_rxdata_fft[k]);
	}
	hipStreamDestroy(ul_cu[i].stream_ul);
	hipStreamDestroy(dl_cu[i].stream_dl);
	free(ul_cu[i].d_rxdata);
	free(ul_cu[i].d_rxdata_fft);
	free(ul_cu[i].d_rxdataF);
	hipHostFree(dl_cu[i].h_txdata);
    hipFree(dl_cu[i].d_txdata);
    hipFree(dl_cu[i].d_txdata_o);
    hipFree(dl_cu[i].d_txdata_ifft);
	hipfftDestroy(dl_cu[i].ifft);
  }

  printf("end cuda\n");
}












